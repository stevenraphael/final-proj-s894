#include "hip/hip_runtime.h"
#include <chrono>
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <filesystem>
#include <fstream>
#include <iostream>
#include <random>
#include <vector>

//#include <cuda/std/unordered_map>
#include <cub/device/device_segmented_reduce.cuh>


////////////////////////////////////////////////////////////////////////////////
// Utility Functions

void cuda_check(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        std::cerr << "CUDA error at " << file << ":" << line << ": "
                  << hipGetErrorString(code) << std::endl;
        exit(1);
    }
}

#define CUDA_CHECK(x) \
    do { \
        cuda_check((x), __FILE__, __LINE__); \
    } while (0)

class GpuMemoryPool {
  public:
    GpuMemoryPool() = default;

    ~GpuMemoryPool();

    GpuMemoryPool(GpuMemoryPool const &) = delete;
    GpuMemoryPool &operator=(GpuMemoryPool const &) = delete;
    GpuMemoryPool(GpuMemoryPool &&) = delete;
    GpuMemoryPool &operator=(GpuMemoryPool &&) = delete;

    void *alloc(size_t size);
    void reset();

  private:
    std::vector<void *> allocations_;
    std::vector<size_t> capacities_;
    size_t next_idx_ = 0;
};



enum class Mode {
    TEST,
    BENCHMARK,
};

template <typename T> struct GpuBuf {
    T *data;

    explicit GpuBuf(size_t n) { CUDA_CHECK(hipMalloc(&data, n * sizeof(T))); }

    explicit GpuBuf(std::vector<T> const &host_data) {
        CUDA_CHECK(hipMalloc(&data, host_data.size() * sizeof(T)));
        CUDA_CHECK(hipMemcpy(
            data,
            host_data.data(),
            host_data.size() * sizeof(T),
            hipMemcpyHostToDevice));
    }

    ~GpuBuf() { CUDA_CHECK(hipFree(data)); }
};


struct Scene {
    int32_t dims;
    int32_t n_points;
    int32_t n_centroids;
    std::vector<float> true_centroids;
    std::vector<float> initial_centroids;
    std::vector<float> features;
};


struct AddOp
{
    template <typename T>
    __device__ __forceinline__
    T operator()(const T &a, const T &b) const {
        return a+b;
    }
};

AddOp add_op;


namespace kmeans {


const int warp_size = 32;
const int block_size = 4;

size_t get_workspace_size(size_t n) {
    return n;
}

__global__ void compute_clusters(
    int n,
    int k,
    int d,
    float *points,
    float *centroids,
    uint32_t *centroid_map
){
    int best_centroid = 0;
    float curr_dist = 0;

    

    int point_idx = threadIdx.x+threadIdx.y*warp_size+blockIdx.x*warp_size*block_size;
    if(point_idx>=n) return;
    for(int idx=0;idx<d;idx++){
        float point_coord = points[point_idx*d+idx];
        float centroid_coord = centroids[idx];
        curr_dist += (point_coord-centroid_coord)*(point_coord-centroid_coord);
    }
    for(int i=1;i<k;i++){    
        float next_dist = 0;
        for(int idx=0;idx<d;idx++){
            float point_coord = points[point_idx*d+idx];
            float centroid_coord = centroids[i*k+idx];
            next_dist += (point_coord-centroid_coord)*(point_coord-centroid_coord);
        }
        if(next_dist<curr_dist){
            curr_dist = next_dist;
            best_centroid = i;
        }
    }

    centroid_map[point_idx] = best_centroid;
}


const int points_per_thread = 8;

const int warp_size_2 = 4;

const int block_size_2 = 32;


const int MAX_CENTROIDS = 100;





__global__ void compute_centroids(
    int n,
    int k,
    int d,
    float *points,
    float *centroids,
    uint32_t *centroid_map,
    float *global_dist_sums,
    int *global_point_counts
){
    int dim = threadIdx.x;
    int point_idx = threadIdx.y*points_per_thread+threadIdx.z*points_per_thread*warp_size_2
                    +blockIdx.x*points_per_thread*warp_size_2*block_size_2;


    int output_idx = point_idx/points_per_thread;


    

    
    float sum_map[MAX_CENTROIDS];
    int count_map[MAX_CENTROIDS];

    for(int i=0;i<k;i++){
        sum_map[i]=0.0;
        count_map[i]=0;
    }
    //cuda::std::unordered_map<int, float> sum_map;
    //cuda::std::unordered_map<int, int> count_map;

    for(int p=point_idx;p<point_idx+points_per_thread;p++){
        if(p>=n) break;
        //int label = centroid_map[p];
        /*if(sum_map.contains(centroid_map[p])){
            if(dim==0)
                count_map[p]++;
            sum_map[p] += points[p*d+dim];
        }*/
        //else{
            if(dim==0)
                count_map[p]+=1;
            sum_map[p] += points[p*d+dim];
        //}
    }
    //__syncthreads();
    for(int i=0;i<k;i++){
        //if(sum_map.contains(i)){
            if(dim==0){
                global_point_counts[((n/points_per_thread)+1)*i+output_idx] = count_map[i];
            }
            global_dist_sums[((n/points_per_thread)+1)*(i*d+dim)+output_idx] = sum_map[i];
        //}
    }

    
}


__global__ void reset_centroids(
    int n,
    int k,
    int d,
    int *point_counts,
    float *centroid_sums,
    float *initial_centroids
){
    for(int c=0;c<k;c++){
        for(int dim=0;dim<d;dim++){
            initial_centroids[c*d+dim]=centroid_sums[c*d+dim]/point_counts[c];
        }
    }
}



void launch_kmeans(
    int n,
    int k,
    int d,
    float *points,
    float *initial_centroids,
    float *output_centroids,
    int *count_offsets_gpu,
    int *sum_offsets_gpu,
    int *total_counts_gpu,
    float *total_sums_gpu,
    GpuMemoryPool &memory_pool
){


    uint32_t *centroid_map = reinterpret_cast<uint32_t *>(memory_pool.alloc(n*sizeof(uint32_t)));
    int *point_counts = reinterpret_cast<int *>(memory_pool.alloc(n*sizeof(int)/points_per_thread));
    float *dist_sums = reinterpret_cast<float *>(memory_pool.alloc(n*d*sizeof(float)/points_per_thread));

    void* d_temp_storage      = nullptr;
    size_t temp_count_bytes = 0;

    int initial_value = 0;

    

    hipcub::DeviceSegmentedReduce::Reduce(
    d_temp_storage,
    temp_count_bytes,
    point_counts,
    total_counts_gpu,
    k,
    count_offsets_gpu,
    count_offsets_gpu + 1,
    add_op,
    initial_value);

    size_t temp_sum_bytes = 0;
    hipcub::DeviceSegmentedReduce::Reduce(
    d_temp_storage,
    temp_sum_bytes,
    dist_sums,
    total_sums_gpu,
    k*d,
    sum_offsets_gpu,
    sum_offsets_gpu + 1,
    add_op,
    initial_value);


    uint8_t *temp_count_storage = reinterpret_cast<uint8_t *>(memory_pool.alloc(temp_count_bytes));
    uint8_t *temp_sum_storage = reinterpret_cast<uint8_t *>(memory_pool.alloc(temp_sum_bytes));


    // step 1: get clusters of points
    // step 2: get local point counts and point sums

    // step 3: reduce point counts and sums

    //https://nvidia.github.io/cccl/cub/api/structcub_1_1DeviceReduce.html#_CPPv4N3cub12DeviceReduceE

    //https://nvidia.github.io/cccl/cub/api/structcub_1_1DeviceSegmentedReduce.html

    //void* d_temp_storage      = nullptr;
    //size_t temp_storage_bytes = 0;


    for(int i=0;i<100;i++){
        dim3 thread_dims_1 = dim3(warp_size, block_size);

        dim3 thread_dims_2 = dim3(d,warp_size_2, block_size_2);

        int num_blocks_2 = n/warp_size_2/block_size_2/points_per_thread+1;

        compute_clusters<<<n/warp_size/block_size+1,thread_dims_1>>>(n,k,d,points,initial_centroids,centroid_map);
        compute_centroids<<<num_blocks_2,thread_dims_2>>>(n,k,d,points,initial_centroids,centroid_map,dist_sums,point_counts);

        hipcub::DeviceSegmentedReduce::Reduce(
        temp_count_storage,
        temp_count_bytes,
        point_counts,
        total_counts_gpu,
        k,
        count_offsets_gpu,
        count_offsets_gpu + 1,
        add_op,
        initial_value);

        hipcub::DeviceSegmentedReduce::Reduce(
        temp_sum_storage,
        temp_sum_bytes,
        dist_sums,
        total_sums_gpu,
        k*d,
        sum_offsets_gpu,
        sum_offsets_gpu + 1,
        add_op,
        initial_value);

        reset_centroids<<<1,1>>>(n,k,d,total_counts_gpu,total_sums_gpu,initial_centroids);
    }

}


}




GpuMemoryPool::~GpuMemoryPool() {
    for (auto ptr : allocations_) {
        CUDA_CHECK(hipFree(ptr));
    }
}

void *GpuMemoryPool::alloc(size_t size) {
    if (next_idx_ < allocations_.size()) {
        auto idx = next_idx_++;
        if (size > capacities_.at(idx)) {
            CUDA_CHECK(hipFree(allocations_.at(idx)));
            CUDA_CHECK(hipMalloc(&allocations_.at(idx), size));
            CUDA_CHECK(hipMemset(allocations_.at(idx), 0, size));
            capacities_.at(idx) = size;
        }
        return allocations_.at(idx);
    } else {
        void *ptr;
        CUDA_CHECK(hipMalloc(&ptr, size));
        CUDA_CHECK(hipMemset(ptr, 0, size));
        allocations_.push_back(ptr);
        capacities_.push_back(size);
        next_idx_++;
        return ptr;
    }
}

void GpuMemoryPool::reset() {
    next_idx_ = 0;
    for (int32_t i = 0; i < allocations_.size(); i++) {
        CUDA_CHECK(hipMemset(allocations_.at(i), 0, capacities_.at(i)));
    }
}

template <typename Reset, typename F>
double benchmark_ms(double target_time_ms, Reset &&reset, F &&f) {
    double best_time_ms = std::numeric_limits<double>::infinity();
    double elapsed_ms = 0.0;
    while (elapsed_ms < target_time_ms) {
        reset();
        CUDA_CHECK(hipDeviceSynchronize());
        auto start = std::chrono::high_resolution_clock::now();
        f();
        CUDA_CHECK(hipDeviceSynchronize());
        auto end = std::chrono::high_resolution_clock::now();
        double this_ms = std::chrono::duration<double, std::milli>(end - start).count();
        elapsed_ms += this_ms;
        best_time_ms = std::min(best_time_ms, this_ms);
    }
    return best_time_ms;
}



struct Results {
    float average_squared_dist;
    std::vector<float> centroids;
    double time_ms;
};


Results run_config(Mode mode, Scene &scene) {
    auto points_gpu = GpuBuf<float>(scene.features);
    auto centroids_gpu = GpuBuf<float>(scene.initial_centroids);
    //auto initial_centroids_gpu = GpuBuf<float>(scene.initial_centroids);
    auto memory_pool = GpuMemoryPool();


    int num_count_segments = scene.n_centroids;
    int num_sum_segments = scene.n_centroids * scene.dims;

    int segment_size = ((scene.n_points/kmeans::points_per_thread)+1);

    std::vector<int> count_offsets;
    std::vector<int> sum_offsets;
    for(int i=0;i<num_count_segments+1;i++){
        count_offsets.push_back(i*segment_size);
    }
    for(int i=0;i<num_sum_segments+1;i++){
        sum_offsets.push_back(i*segment_size);
    }


    std::vector<int> total_counts(num_count_segments);
    std::vector<float> total_sums(num_sum_segments);


    auto count_offsets_gpu = GpuBuf<int>(count_offsets);
    auto sum_offsets_gpu = GpuBuf<int>(sum_offsets);

    auto total_counts_gpu = GpuBuf<int>(total_counts);
    auto total_sums_gpu = GpuBuf<float>(total_sums);

    


    auto reset = [&]() {
        /*CUDA_CHECK(
            hipMemset(points_gpu.data, 0, scene.features.size() * sizeof(float)));
        CUDA_CHECK(
            hipMemset(centroids_gpu.data, 0, scene.true_centroids.size() * sizeof(float)));*/
        CUDA_CHECK(
            hipMemset(count_offsets_gpu.data, 0, (num_count_segments+1) * sizeof(int)));
        CUDA_CHECK(
            hipMemset(sum_offsets_gpu.data, 0, (num_sum_segments+1) * sizeof(int)));
        CUDA_CHECK(
            hipMemset(total_counts_gpu.data, 0, (num_count_segments) * sizeof(int)));
        CUDA_CHECK(
            hipMemset(total_sums_gpu.data, 0, (num_sum_segments) * sizeof(float)));

        memory_pool.reset();
    };

    auto f = [&]() {
        kmeans::launch_kmeans(
            scene.n_points,
            scene.n_centroids,
            scene.dims,
            points_gpu.data,
            centroids_gpu.data,
            centroids_gpu.data,
            count_offsets_gpu.data,
            sum_offsets_gpu.data,
            total_counts_gpu.data,
            total_sums_gpu.data,
            memory_pool);
    };

    reset();
    f();


    auto returned_centroids = std::vector<float>(scene.initial_centroids.size(), 0.0f);
     CUDA_CHECK(hipMemcpy(
        returned_centroids.data(),
        centroids_gpu.data,
        scene.initial_centroids.size() * sizeof(float),
        hipMemcpyDeviceToHost));

    float squared_dist_sum = 0;

    //printf("%d %d %d", scene.true_centroids.size(),scene.n_centroids, scene.dims);

    for(int i=0;i<scene.n_centroids;i++){
        for(int j=0;j<scene.dims;j++){
            squared_dist_sum += (scene.true_centroids[i*scene.dims+j]-returned_centroids[i*scene.dims+j]);
                                //* (scene.true_centroids[i*scene.dims+j]-returned_centroids[i*scene.dims+j]);
        }
    }

    float average_squared_dist = squared_dist_sum/scene.n_centroids;

    double time_ms = 0.0;//= benchmark_ms(1000.0, reset, f);

    return Results{
        average_squared_dist,
        std::move(returned_centroids),
        time_ms,
    };

}


template <typename Rng>
Scene gen_random(Rng &rng, int32_t dims, int32_t n_points, int32_t n_centroids){
    auto unif_100 = std::uniform_real_distribution<float>(-100.0f, 100.0f);
    auto unif_0_1 = std::uniform_real_distribution<float>(0.0f, 1.0f);
    auto true_centroids = std::vector<float>();

    const float stddev = 10.0;

    for (int32_t i = 0; i < n_centroids*dims; i++) {
        float z;
        z = unif_0_1(rng);

    
        // float z = std::max(unif_0_1(rng), unif_0_1(rng));
        true_centroids.push_back(z);
    }

    auto normal = std::normal_distribution<double>(0.0, stddev);

    auto features = std::vector<float>();

    for (int32_t cent = 0; cent < n_centroids; cent++) {
        for(int point=0;point<1+n_points/n_centroids;point++){
            if((1+n_points/n_centroids)*cent+point>=n_points){
                break;
            }
            for(int dim=0;dim<dims;dim++){
                float feature = normal(rng)+true_centroids[cent*dims+dim];
                features.push_back(feature);
            }
        }
    }

    auto initial_centroids = std::vector<float>();
    for (int32_t i = 0; i < n_centroids*dims; i++) {
        float z;
        z = unif_0_1(rng);

    
        // float z = std::max(unif_0_1(rng), unif_0_1(rng));
        initial_centroids.push_back(z);
    }

    auto scene = Scene{dims, n_points, n_centroids, true_centroids, initial_centroids, features};

    return scene;

}


struct SceneTest {
    std::string name;
    Mode mode;
    Scene scene;
};

int main(int argc, char const *const *argv) {
    auto rng = std::mt19937(0xCA7CAFE);
    auto scenes = std::vector<SceneTest>();
    scenes.push_back(
        {"test1", Mode::BENCHMARK, gen_random(rng, 10, 65536, 10)});
    int32_t fail_count = 0;

    int32_t count = 0;
    for (auto &scene_test : scenes) {
        auto i = count++;
        printf("\nTesting scene '%s'\n", scene_test.name.c_str());
        auto results = run_config(scene_test.mode, scene_test.scene);
        printf("  Error: %f \n", results.average_squared_dist);
    }

}


