#include "hip/hip_runtime.h"
#include <chrono>
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <filesystem>
#include <fstream>
#include <iostream>
#include <random>
#include <vector>



const int warp_size = 32;
const int block_size = 4;

__device__ void compute_clusters(
    int n,
    int k,
    int d,
    float *points,
    float *centroids,
    uint32_t *centroid_map
){
    int best_centroid = 0;
    float curr_dist = 0;

    

    int point_idx = threadIdx.x+threadIdx.y*warp_size+threadIdx.z*block_size+blockIdx.x*warp_size*block_size;
    for(int idx=0;idx<d;idx++){
        float point_coord = points[point_idx*d+idx];
        float centroid_coord = centroids[idx];
        curr_dist += (point_coord-centroid_coord)*(point_coord-centroid_coord);
    }
    for(int i=1;i<k;i++){    
        for(int idx=0;idx<d;idx++){
            float next_dist = 0;
            float point_coord = points[point_idx*d+idx];
            float centroid_coord = centroids[i*k+idx];
            next_dist += (point_coord-centroid_coord)*(point_coord-centroid_coord);
        }
        if(next_dist<curr_dist){
            curr_dist = next_dist;
            best_centroid = i;
        }
    }

    centroid_map[point_idx] = best_centroid;
}


const int points_per_thread = 1;

__device__ void compute_centroids(
    int n,
    int k,
    int d,
    float *points,
    float *centroids,
    uint32_t *centroid_map,
    float *local_dist_sums,
    float *local_point_counts,
    float *global_dist_sums,
    float *global_point_counts
){
}



__global__ void kmeans(
    int n,
    
    int k,
    int d,
    float *points,
    float *centroids,
    uint32_t *centroid_map,
){
}